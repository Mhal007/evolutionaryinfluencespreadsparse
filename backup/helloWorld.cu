
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

const int N = 16;
const int blocksize = 16;

__global__
void hello(char *a, int *b)
{
	a[threadIdx.x] += b[threadIdx.x];
}

int main()
{
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	printf("%s", a);

    hipError_t err1 = hipMalloc( (void**)&ad, csize );
    hipError_t err2 = hipMalloc( (void**)&bd, isize );
	

    if (err1 != hipSuccess) {
        cout << "Error allocating memory for ad." << endl;
        printf("CUDA error: %s\n", hipGetErrorString(err1));
        return 0;
    }
    if (err2 != hipSuccess) {
        cout << "Error allocating memory for bd." << endl;
        printf("CUDA error: %s\n", hipGetErrorString(err2));
        return 0;
    }
    
	
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
	hipFree( ad );
	hipFree( bd );

	printf("%s\n", a);
	return EXIT_SUCCESS;
}
